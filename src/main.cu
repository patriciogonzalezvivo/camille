#include "hip/hip_runtime.h"

#include <string>
#include <time.h>

#include <hip/hip_runtime.h>
#include <>
#define BLOCK_SIZE 8

#include "hilma/ops/fs.h"

#include "hilma/io/obj.h"
#include "hilma/io/ply.h"
#include "hilma/io/stl.h"
#include "hilma/io/gltf.h"
#include "hilma/io/png.h"

#include "hilma/types/image.h"

#include "hilma/ops/generate.h"
#include "hilma/ops/transform.h"
#include "hilma/ops/raytrace.h"
#include "hilma/ops/intersection.h"
#include "hilma/ops/compute.h"
#include "hilma/ops/image.h"

#include "lygia/math/make.cuh"
#include "lygia/math/cross.cuh"
#include "lygia/math/clamp.cuh"
#include "lygia/math/floor.cuh"
#include "lygia/math/length.cuh"
#include "lygia/math/normalize.cuh"
#include "lygia/math/operations.cuh"
#include "lygia/geometry/aabb.cuh"
#include "lygia/geometry/triangle.cuh"

__global__ void kernel( Triangle* _tris, float3* _trisNormals, int _Ntris, 
                        float *_pixels, 
                        AABB _aabb, float3 _bdiagonal, float _max_dist, 
                        float _voxel_size, int _voxel_resolution, int _layersTotal)  {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= _voxel_resolution || y >= _voxel_resolution || z >= _voxel_resolution)
        return;

    float3 p = make_float3((float)x, (float)y, (float)z) * _voxel_size;
    p = _aabb.min + p * _bdiagonal;

    float min_dist = 99999.9f;
    for (int i = 0; i < _Ntris; i++ ) {
        float dist = signedDistance(_tris[i], _trisNormals[i], p);
        if (abs(dist) < abs(min_dist) )
            min_dist = dist;
    }
    min_dist = min_dist/_max_dist;
    min_dist = clamp(min_dist, -1.0f, 1.0f) * 0.5 + 0.5;

    int layerX = (z % _layersTotal) * _voxel_resolution; 
    int layerY = (z / _layersTotal) * _voxel_resolution;

    int width = _voxel_resolution * _layersTotal;
    int index = (layerX + x) + (layerY + y) * width;
    _pixels[index] = min_dist;
}

int main(int argc, char **argv) {

    std::string filename = std::string(argv[1]);
    std::string ext = hilma::getExt(filename);

    hilma::Mesh mesh;
    if ( ext == "ply" || ext == "PLY" )
        hilma::loadPly( filename, mesh );

    else if ( ext == "obj" || ext == "OBJ" )
        hilma::loadObj( filename, mesh );

    else if ( ext == "stl" || ext == "STL" )
        hilma::loadStl( filename, mesh );

    else if (   ext == "gltf" || ext == "GLTF" ||
                ext == "glb" || ext == "GLB" )
        hilma::loadGltf( filename, mesh );

    std::cout << "Mesh loaded" << std::endl;

    clock_t start, end;
    start = clock();

    hilma::center(mesh);
    hilma::BoundingBox bbox = getBoundingBox(mesh);
    bbox.square();
    std::vector<hilma::Triangle> mesh_triangles = mesh.getTriangles();

    // Create array triangles and axis aligned bounding box 
    Triangle*   cpuTris = new Triangle[mesh_triangles.size()];
    float3*     cpuTrisNormals = new float3[mesh_triangles.size()];

    AABB aabb;
    aabb.min = make_float3(99999.9f);
    aabb.max = make_float3(-99999.9f);

    for (size_t i = 0; i < mesh_triangles.size(); i++) {
        cpuTris[i].a = make_float3(mesh_triangles[i][0]);
        cpuTris[i].b = make_float3(mesh_triangles[i][1]);
        cpuTris[i].c = make_float3(mesh_triangles[i][2]);
        cpuTrisNormals[i] = normal(cpuTris[i]);
        expand( aabb, cpuTris[i] );
    }

    // square the aabb on the longest side
    square(aabb);

    Triangle* gpuTris;
    hipMalloc(&gpuTris, sizeof(Triangle) * mesh_triangles.size());
    hipMemcpy(gpuTris, cpuTris, sizeof(Triangle) * mesh_triangles.size(), hipMemcpyHostToDevice);

    float3* gpuTrisNormals;
    hipMalloc(&gpuTrisNormals, sizeof(Triangle) * mesh_triangles.size());
    hipMemcpy(gpuTrisNormals, cpuTrisNormals, sizeof(float3) * mesh_triangles.size(), hipMemcpyHostToDevice);

    // expand it just a bit
    float        paddingPct = 0.001f;
    float3        bdiagonal = diagonal(aabb);
    float          max_dist = length(bdiagonal);
    expand( aabb, (max_dist*max_dist) * paddingPct );
    max_dist *= 0.5f;

    // Calculate the voxel matrix resolution
    int          resolution = 8;
    int    voxel_resolution = std::pow(2, resolution);
    float        voxel_size = 1.0/float(voxel_resolution);
    int         layersTotal = std::sqrt(voxel_resolution);

    // Calculate the image resolution and allocate pixels
    int    image_resolution = voxel_resolution * layersTotal;
    int      image_channels = 1;
    float *image_pixels;
    hipMalloc(&image_pixels, image_channels * image_resolution * image_resolution * sizeof(float));

    // define Kernel
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(voxel_resolution / threads.x + 1, 
                voxel_resolution / threads.y + 1, 
                voxel_resolution / threads.z + 1);

    // run the kernel
    kernel<<<blocks, threads>>>(gpuTris, gpuTrisNormals, (int)mesh_triangles.size(), image_pixels, aabb, bdiagonal, max_dist, voxel_size, voxel_resolution, layersTotal);

    // free(cpuTris);
    hipFree(gpuTris);
    hilma::Image sdf = hilma::Image(image_resolution, image_resolution, image_channels);
    hipMemcpy(&sdf[0], image_pixels, image_channels * image_resolution * image_resolution * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(image_pixels);
    
    end = clock();
    double duration_sec = double(end-start)/CLOCKS_PER_SEC;

    std::cout << "Took " << duration_sec << "secs" << std::endl;

    // Save Image
    filename.erase(filename.length() - ext.length());
    filename += "png";

    hilma::flip(sdf);
    hilma::savePng(filename, sdf);

    return 1;
}
